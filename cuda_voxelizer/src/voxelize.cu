#include "hip/hip_runtime.h"
#include "voxelize.cuh"

// CUDA Global Memory variables

// Debug counters for some sanity checks
#ifdef _DEBUG
__device__ size_t debug_d_n_voxels_marked = 0;
__device__ size_t debug_d_n_triangles = 0;
__device__ size_t debug_d_n_voxels_tested = 0;
#endif

// Set a bit in the giant voxel table. This involves doing an atomic operation on a 32-bit word in memory.
// Blocking other threads writing to it for a very short time
__device__ __inline__ void setBit(unsigned int* voxel_table, size_t index){
	size_t int_location = index / size_t(32);
	unsigned int bit_pos = size_t(31) - (index % size_t(32)); // we count bit positions RtL, but array indices LtR
	unsigned int mask = 1 << bit_pos;
	atomicOr(&(voxel_table[int_location]), mask);
}

__device__ float3 calculateBarycentric(float3 v0, float3 v1, float3 v2, float3 p) {
    float3 v0v1 = v1 - v0;
    float3 v0v2 = v2 - v0;
    float3 v0p = p - v0;
    float d00 = dot(v0v1, v0v1);
    float d01 = dot(v0v1, v0v2);
    float d11 = dot(v0v2, v0v2);
    float d20 = dot(v0p, v0v1);
    float d21 = dot(v0p, v0v2);
    float denom = d00 * d11 - d01 * d01;
    float v = (d11 * d20 - d01 * d21) / denom;
    float w = (d00 * d21 - d01 * d20) / denom;
    float u = 1.0f - v - w;
    return make_float3(u, v, w);
}

__device__ float3 perspectiveDivide(float3 v) {
    return make_float3(v.x / v.z, v.y / v.z, 1.0f / v.z);
}

__device__ float2 interpolateUV(float2 uv0, float2 uv1, float2 uv2, float3 barycentric, float w0, float w1, float w2) {
    float2 uv = make_float2(
        (uv0.x * barycentric.x / w0 + uv1.x * barycentric.y / w1 + uv2.x * barycentric.z / w2) / (barycentric.x / w0 + barycentric.y / w1 + barycentric.z / w2),
        (uv0.y * barycentric.x / w0 + uv1.y * barycentric.y / w1 + uv2.y * barycentric.z / w2) / (barycentric.x / w0 + barycentric.y / w1 + barycentric.z / w2)
    );
    return uv;
}

__device__ uchar4 bilinearSample(uchar4* texture_data, float2 uv, int texture_width, int texture_height) {
    float u = uv.x * (texture_width - 1);
    float v = (1.0f - uv.y) * (texture_height - 1);  // Flip v coordinate
    
    int x0 = floorf(u);
    int y0 = floorf(v);
    int x1 = min(x0 + 1, texture_width - 1);
    int y1 = min(y0 + 1, texture_height - 1);
    
    float dx = u - x0;
    float dy = v - y0;
    
    uchar4 c00 = texture_data[y0 * texture_width + x0];
    uchar4 c10 = texture_data[y0 * texture_width + x1];
    uchar4 c01 = texture_data[y1 * texture_width + x0];
    uchar4 c11 = texture_data[y1 * texture_width + x1];
    
    uchar4 color;
    color.x = (1 - dx) * (1 - dy) * c00.x + dx * (1 - dy) * c10.x + (1 - dx) * dy * c01.x + dx * dy * c11.x;
    color.y = (1 - dx) * (1 - dy) * c00.y + dx * (1 - dy) * c10.y + (1 - dx) * dy * c01.y + dx * dy * c11.y;
    color.z = (1 - dx) * (1 - dy) * c00.z + dx * (1 - dy) * c10.z + (1 - dx) * dy * c01.z + dx * dy * c11.z;
    color.w = 255;  // Assuming full opacity
    
    return color;
}

// Main triangle voxelization method
__global__ void voxelize_triangle(voxinfo info, float* triangle_data, float* uv_data, unsigned int* voxel_table, uchar4* color_table, uchar4* texture_data, int texture_width, int texture_height, bool morton_order){
    size_t thread_id = threadIdx.x + blockIdx.x * blockDim.x;
    size_t stride = blockDim.x * gridDim.x;

    // Common variables used in the voxelization process
    float3 delta_p = make_float3(info.unit.x, info.unit.y, info.unit.z);
    int3 grid_max = make_int3(info.gridsize.x - 1, info.gridsize.y - 1, info.gridsize.z - 1); // grid max (grid runs from 0 to gridsize-1)

    while (thread_id < info.n_triangles){ // every thread works on specific triangles in its stride
        size_t t = thread_id * 9; // triangle contains 9 vertices
        size_t uv_t = thread_id * 6; // triangle contains 6 uv coordinates

        // COMPUTE COMMON TRIANGLE PROPERTIES
        // Move vertices to origin using bbox
        float3 v0 = make_float3(triangle_data[t], triangle_data[t + 1], triangle_data[t + 2]) - info.bbox.min;
        float3 v1 = make_float3(triangle_data[t + 3], triangle_data[t + 4], triangle_data[t + 5]) - info.bbox.min;
        float3 v2 = make_float3(triangle_data[t + 6], triangle_data[t + 7], triangle_data[t + 8]) - info.bbox.min;

        // UV coordinates
        float2 uv0 = make_float2(uv_data[uv_t], uv_data[uv_t + 1]);
        float2 uv1 = make_float2(uv_data[uv_t + 2], uv_data[uv_t + 3]);
        float2 uv2 = make_float2(uv_data[uv_t + 4], uv_data[uv_t + 5]);

        // Edge vectors
        float3 e0 = v1 - v0;
        float3 e1 = v2 - v1;
        float3 e2 = v0 - v2;
        // Normal vector pointing up from the triangle
        float3 n = normalize(cross(e0, e1));

        // COMPUTE TRIANGLE BBOX IN GRID
        // Triangle bounding box in world coordinates is min(v0,v1,v2) and max(v0,v1,v2)
        AABox<float3> t_bbox_world(fminf(v0, fminf(v1, v2)), fmaxf(v0, fmaxf(v1, v2)));
        // Triangle bounding box in voxel grid coordinates is the world bounding box divided by the grid unit vector
        AABox<int3> t_bbox_grid;
        t_bbox_grid.min = clamp(float3_to_int3(t_bbox_world.min / info.unit), make_int3(0, 0, 0), grid_max);
        t_bbox_grid.max = clamp(float3_to_int3(t_bbox_world.max / info.unit), make_int3(0, 0, 0), grid_max);

        // PREPARE PLANE TEST PROPERTIES
        float3 c = make_float3(0.0f, 0.0f, 0.0f);
        if (n.x > 0.0f) { c.x = info.unit.x; }
        if (n.y > 0.0f) { c.y = info.unit.y; }
        if (n.z > 0.0f) { c.z = info.unit.z; }
        float d1 = dot(n, (c - v0));
        float d2 = dot(n, ((delta_p - c) - v0));

        // PREPARE PROJECTION TEST PROPERTIES
        // XY plane
        float2 n_xy_e0 = make_float2(-1.0f * e0.y, e0.x);
        float2 n_xy_e1 = make_float2(-1.0f * e1.y, e1.x);
        float2 n_xy_e2 = make_float2(-1.0f * e2.y, e2.x);
        if (n.z < 0.0f) {
            n_xy_e0 = -n_xy_e0;
            n_xy_e1 = -n_xy_e1;
            n_xy_e2 = -n_xy_e2;
        }
        float d_xy_e0 = (-1.0f * dot(n_xy_e0, make_float2(v0.x, v0.y))) + max(0.0f, info.unit.x * n_xy_e0.x) + max(0.0f, info.unit.y * n_xy_e0.y);
        float d_xy_e1 = (-1.0f * dot(n_xy_e1, make_float2(v1.x, v1.y))) + max(0.0f, info.unit.x * n_xy_e1.x) + max(0.0f, info.unit.y * n_xy_e1.y);
        float d_xy_e2 = (-1.0f * dot(n_xy_e2, make_float2(v2.x, v2.y))) + max(0.0f, info.unit.x * n_xy_e2.x) + max(0.0f, info.unit.y * n_xy_e2.y);
        // YZ plane
        float2 n_yz_e0 = make_float2(-1.0f * e0.z, e0.y);
        float2 n_yz_e1 = make_float2(-1.0f * e1.z, e1.y);
        float2 n_yz_e2 = make_float2(-1.0f * e2.z, e2.y);
        if (n.x < 0.0f) {
            n_yz_e0 = -n_yz_e0;
            n_yz_e1 = -n_yz_e1;
            n_yz_e2 = -n_yz_e2;
        }
        float d_yz_e0 = (-1.0f * dot(n_yz_e0, make_float2(v0.y, v0.z))) + max(0.0f, info.unit.y * n_yz_e0.x) + max(0.0f, info.unit.z * n_yz_e0.y);
        float d_yz_e1 = (-1.0f * dot(n_yz_e1, make_float2(v1.y, v1.z))) + max(0.0f, info.unit.y * n_yz_e1.x) + max(0.0f, info.unit.z * n_yz_e1.y);
        float d_yz_e2 = (-1.0f * dot(n_yz_e2, make_float2(v2.y, v2.z))) + max(0.0f, info.unit.y * n_yz_e2.x) + max(0.0f, info.unit.z * n_yz_e2.y);
        // ZX plane
        float2 n_zx_e0 = make_float2(-1.0f * e0.x, e0.z);
        float2 n_zx_e1 = make_float2(-1.0f * e1.x, e1.z);
        float2 n_zx_e2 = make_float2(-1.0f * e2.x, e2.z);
        if (n.y < 0.0f) {
            n_zx_e0 = -n_zx_e0;
            n_zx_e1 = -n_zx_e1;
            n_zx_e2 = -n_zx_e2;
        }
        float d_xz_e0 = (-1.0f * dot(n_zx_e0, make_float2(v0.z, v0.x))) + max(0.0f, info.unit.x * n_zx_e0.x) + max(0.0f, info.unit.z * n_zx_e0.y);
        float d_xz_e1 = (-1.0f * dot(n_zx_e1, make_float2(v1.z, v1.x))) + max(0.0f, info.unit.x * n_zx_e1.x) + max(0.0f, info.unit.z * n_zx_e1.y);
        float d_xz_e2 = (-1.0f * dot(n_zx_e2, make_float2(v2.z, v2.x))) + max(0.0f, info.unit.x * n_zx_e2.x) + max(0.0f, info.unit.z * n_zx_e2.y);

        // test possible grid boxes for overlap
        for (int z = t_bbox_grid.min.z; z <= t_bbox_grid.max.z; z++){
            for (int y = t_bbox_grid.min.y; y <= t_bbox_grid.max.y; y++){
                for (int x = t_bbox_grid.min.x; x <= t_bbox_grid.max.x; x++){
                    // if (checkBit(voxel_table, location)){ continue; }
#ifdef _DEBUG
                    atomicAdd(&debug_d_n_voxels_tested, 1);
#endif
                    // TRIANGLE PLANE THROUGH BOX TEST
                    float3 p = make_float3(x * info.unit.x, y * info.unit.y, z * info.unit.z);
                    float nDOTp = dot(n, p);
                    if (((nDOTp + d1) * (nDOTp + d2)) > 0.0f) { continue; }

                    // PROJECTION TESTS
                        // XY
                    float2 p_xy = make_float2(p.x, p.y);
                    if ((dot(n_xy_e0, p_xy) + d_xy_e0) < 0.0f) { continue; }
                    if ((dot(n_xy_e1, p_xy) + d_xy_e1) < 0.0f) { continue; }
                    if ((dot(n_xy_e2, p_xy) + d_xy_e2) < 0.0f) { continue; }

                    // YZ
                    float2 p_yz = make_float2(p.y, p.z);
                    if ((dot(n_yz_e0, p_yz) + d_yz_e0) < 0.0f) { continue; }
                    if ((dot(n_yz_e1, p_yz) + d_yz_e1) < 0.0f) { continue; }
                    if ((dot(n_yz_e2, p_yz) + d_yz_e2) < 0.0f) { continue; }

                    // XZ	
                    float2 p_zx = make_float2(p.z, p.x);
                    if ((dot(n_zx_e0, p_zx) + d_xz_e0) < 0.0f) { continue; }
                    if ((dot(n_zx_e1, p_zx) + d_xz_e1) < 0.0f) { continue; }
                    if ((dot(n_zx_e2, p_zx) + d_xz_e2) < 0.0f) { continue; }

#ifdef _DEBUG
                    atomicAdd(&debug_d_n_voxels_marked, 1);
#endif
                    // Calculate perspective-correct barycentric coordinates
                    float3 bary = calculateBarycentric(v0, v1, v2, p);
                    float w0 = 1.0f / dot(bary, make_float3(1.0f / v0.z, 1.0f / v1.z, 1.0f / v2.z));
                    float w1 = w0 * bary.y / v1.z;
                    float w2 = w0 * bary.z / v2.z;
                    w0 = w0 * bary.x / v0.z;
                    
                    // Interpolate UV coordinates
                    float2 uv = interpolateUV(uv0, uv1, uv2, bary, w0, w1, w2);
                    
                    // Clamp UV coordinates instead of wrapping
                    uv.x = fmaxf(0.0f, fminf(1.0f, uv.x));
                    uv.y = fmaxf(0.0f, fminf(1.0f, uv.y));
                    
                    // Sample texture using bilinear interpolation
                    uchar4 color = bilinearSample(texture_data, uv, texture_width, texture_height);
                    
                    // Set voxel and assign color
                    if (morton_order) {
                        size_t location = mortonEncode_LUT(x, y, z);
                        setBit(voxel_table, location);
                    } else {
                        size_t location = 
                            static_cast<size_t>(x) + 
                            (static_cast<size_t>(y) * static_cast<size_t>(info.gridsize.x)) + 
                            (static_cast<size_t>(z) * (static_cast<size_t>(info.gridsize.y) * static_cast<size_t>(info.gridsize.x)));
                        setBit(voxel_table, location);
                    }
                    
                    size_t voxel_idx = x + (y * info.gridsize.x) + (z * info.gridsize.x * info.gridsize.y);
                    color_table[voxel_idx] = color;
                }
            }
        }
#ifdef _DEBUG
        atomicAdd(&debug_d_n_triangles, 1);
#endif
        thread_id += stride;
    }
}


void voxelize(const voxinfo& v, float* triangle_data, float* uv_data, unsigned int* vtable, uchar4* color_table, uchar4* texture_data, int texture_width, int texture_height, bool morton_code) {
    float elapsedTime;

    // Create timers, set start time
    hipEvent_t start_vox, stop_vox;
    checkCudaErrors(hipEventCreate(&start_vox));
    checkCudaErrors(hipEventCreate(&stop_vox));

    // Copy morton LUT if we're encoding to morton
    if (morton_code){
        checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(morton256_x), host_morton256_x, 256 * sizeof(uint32_t)));
        checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(morton256_y), host_morton256_y, 256 * sizeof(uint32_t)));
        checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(morton256_z), host_morton256_z, 256 * sizeof(uint32_t)));
    }

    // Estimate best block and grid size using CUDA Occupancy Calculator
    int blockSize;   // The launch configurator returned block size 
    int minGridSize; // The minimum grid size needed to achieve the  maximum occupancy for a full device launch 
    int gridSize;    // The actual grid size needed, based on input size 
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, voxelize_triangle, 0, 0);
    // Round up according to array size 
    gridSize = static_cast<int>((v.n_triangles + blockSize - 1) / blockSize);

    checkCudaErrors(hipEventRecord(start_vox, 0));
    voxelize_triangle<<<gridSize, blockSize>>>(v, triangle_data, uv_data, vtable, color_table, texture_data, texture_width, texture_height, morton_code);

    hipDeviceSynchronize();
    checkCudaErrors(hipEventRecord(stop_vox, 0));
    checkCudaErrors(hipEventSynchronize(stop_vox));
    checkCudaErrors(hipEventElapsedTime(&elapsedTime, start_vox, stop_vox));
    printf("[Perf] Voxelization GPU time: %.1f ms\n", elapsedTime);

    // SANITY CHECKS
#ifdef _DEBUG
    size_t debug_n_triangles, debug_n_voxels_marked, debug_n_voxels_tested;
    checkCudaErrors(hipMemcpyFromSymbol((void*)&(debug_n_triangles), debug_d_n_triangles, sizeof(debug_d_n_triangles), 0, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpyFromSymbol((void*)&(debug_n_voxels_marked), debug_d_n_voxels_marked, sizeof(debug_d_n_voxels_marked), 0, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpyFromSymbol((void*)&(debug_n_voxels_tested), debug_d_n_voxels_tested, sizeof(debug_d_n_voxels_tested), 0, hipMemcpyDeviceToHost));
    printf("[Debug] Processed %llu triangles on the GPU \n", debug_n_triangles);
    printf("[Debug] Tested %llu voxels for overlap on GPU \n", debug_n_voxels_tested);
    printf("[Debug] Marked %llu voxels as filled (includes duplicates!) \n", debug_n_voxels_marked);
#endif

    // Destroy timers
    checkCudaErrors(hipEventDestroy(start_vox));
    checkCudaErrors(hipEventDestroy(stop_vox));
}
